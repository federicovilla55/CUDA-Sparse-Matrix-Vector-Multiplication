#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define CHECK(call) \
{ \
  const hipError_t err = call; \
  if (err != hipSuccess) { \
    printf("%s in %s at line %d\n", hipGetErrorString(err), \
    __FILE__, __LINE__); \
    exit(EXIT_FAILURE); \
  } \
}

#define CHECK_KERNELCALL() \
{ \
  const hipError_t err = hipGetLastError(); \
  if (err != hipSuccess) { \
    printf("%s in %s at line %d\n", hipGetErrorString(err), \
    __FILE__, __LINE__); \
    exit(EXIT_FAILURE); \
  } \
}

__global__ void spmv_csr_kernel(int num_rows, int * ptr, int * indices,  
                                       float * data, float * x, float * y){
    	// Warp thread identifier
    	int lane = threadIdx.x & (32-1);

    	// Global Warp index, one row per warp
    	int row = (blockIdx.x * blockDim.x + threadIdx.x) / 32;

        // Shared memory to perform parallel reduction
    	__shared__ volatile float vals[1024];
    
    	if (row < num_rows)
    	{
        	float dotProduct = 0;

            int row_start = ptr[row];
        	int row_end = ptr[row+1];

        	for (int j = row_start + lane; j < row_end; j += 32){
            		dotProduct += data[j] * x[indices[j]];
        	}

        	vals[threadIdx.x] = dotProduct;
        
        	// Parallel Reduction
        	if (lane < 16) vals[threadIdx.x] += vals[threadIdx.x + 16];
        	if (lane <  8) vals[threadIdx.x] += vals[threadIdx.x + 8];
        	if (lane <  4) vals[threadIdx.x] += vals[threadIdx.x + 4];
        	if (lane <  2) vals[threadIdx.x] += vals[threadIdx.x + 2];
        	if (lane <  1) vals[threadIdx.x] += vals[threadIdx.x + 1];
        
        	// First thread in warp writes result
        	if (lane == 0)
        	{
            		y[row] = vals[threadIdx.x];
        	}
    	}	
}


double get_time() // function to get the time of day in seconds
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

// Reads a sparse matrix and represents it using CSR (Compressed Sparse Row) format
void read_matrix(int **row_ptr, int **col_ind, float **values, const char *filename, int *num_rows, int *num_cols, int *num_vals) {
    
    FILE *file = fopen(filename, "r");
    if (file == NULL) {
        fprintf(stdout, "File cannot be opened!\n");
        exit(0);
    }
    
    // Get number of rows, columns, and non-zero values
    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    int *row_ptr_t = (int *) malloc((*num_rows + 1) * sizeof(int));
    int *col_ind_t = (int *) malloc(*num_vals * sizeof(int));
    float *values_t = (float *) malloc(*num_vals * sizeof(float));
    
    // Collect occurances of each row for determining the indices of row_ptr
    int *row_occurances = (int *) malloc(*num_rows * sizeof(int));
    for (int i = 0; i < *num_rows; i++) {
        row_occurances[i] = 0;
    }
    
    int row, column;
    float value;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF) {
        // Subtract 1 from row and column indices to match C format
        row--;
        column--;
        
        row_occurances[row]++;
    }
    
    // Set row_ptr
    int index = 0;
    for (int i = 0; i < *num_rows; i++) {
        row_ptr_t[i] = index;
        index += row_occurances[i];
    }
    row_ptr_t[*num_rows] = *num_vals;
    free(row_occurances);
    
    // Set the file position to the beginning of the file
    rewind(file);
    
    // Read the file again, save column indices and values
    for (int i = 0; i < *num_vals; i++) {
        col_ind_t[i] = -1;
    }
    
    if(fscanf(file, "%d %d %d\n", num_rows, num_cols, num_vals)==EOF)
        printf("Error reading file");
    
    int i = 0;
    while (fscanf(file, "%d %d %f\n", &row, &column, &value) != EOF) {
        row--;
        column--;
        
        // Find the correct index (i + row_ptr_t[row]) using both row information and an index i
        while (col_ind_t[i + row_ptr_t[row]] != -1) {
            i++;
        }
        col_ind_t[i + row_ptr_t[row]] = column;
        values_t[i + row_ptr_t[row]] = value;
        i = 0;
    }
    
    fclose(file);
    
    *row_ptr = row_ptr_t;
    *col_ind = col_ind_t;
    *values = values_t;
}

// CPU implementation of SPMV using CSR, DO NOT CHANGE THIS
void spmv_csr_sw(const int *row_ptr, const int *col_ind, const float *values, const int num_rows, const float *x, float *y) {
    for (int i = 0; i < num_rows; i++) {
        float dotProduct = 0;
        const int row_start = row_ptr[i];
        const int row_end = row_ptr[i + 1];
        
        for (int j = row_start; j < row_end; j++) {
            dotProduct += values[j] * x[col_ind[j]];
        }
        
        y[i] = dotProduct;
    }
}

int main(int argc, const char * argv[]) {

    if (argc != 3) {
        printf("Usage: ./exec matrix_file number_threads\n");
        return 0;
    }
    
    int *row_ptr, *col_ind, num_rows, num_cols, num_vals;
    float *values;
    
    const char *filename = argv[1];
    int BlockDim = atoi(argv[2]);

    double start_cpu, end_cpu;
    
    read_matrix(&row_ptr, &col_ind, &values, filename, &num_rows, &num_cols, &num_vals);
    
    float *x = (float *) malloc(num_rows * sizeof(float));
    float *y_sw = (float *) malloc(num_rows * sizeof(float));
    
    // Generate a random vector

    srand(time(NULL));

    for (int i = 0; i < num_rows; i++) {
        x[i] = (rand()%100)/(rand()%100+1); //the number we use to divide cannot be 0, that's the reason of the +1
    }
    
    // Compute in sw
    start_cpu = get_time();
    spmv_csr_sw(row_ptr, col_ind, values, num_rows, x, y_sw);
    end_cpu = get_time();

    // Print time
    printf("SPMV Time CPU: %.10lf\n", end_cpu - start_cpu);


    int *ptr, *indices;
    float *data, *c_x, *c_y; 

    
    hipMalloc(&ptr, (num_rows+1)*sizeof(int));
    hipMalloc(&indices, num_vals * sizeof(int));
    hipMalloc(&data, num_vals * sizeof(float));
    hipMalloc(&c_x, num_rows * sizeof(float));
    hipMalloc(&c_y, num_rows * sizeof(float));
    
    hipMemcpy(ptr, row_ptr, (num_rows+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(indices, col_ind, num_vals*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(data, values, (num_vals)*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(c_x, x, (num_rows)*sizeof(float), hipMemcpyHostToDevice);
    

    dim3 blocksPerGrid(num_rows/(BlockDim/32), 1, 1);
    dim3 threadsPerBlock(BlockDim, 1, 1);

    start_cpu = get_time();
    spmv_csr_kernel<<<blocksPerGrid, threadsPerBlock>>> (num_rows, ptr, indices, data, c_x, c_y);
    CHECK_KERNELCALL();
    CHECK(hipDeviceSynchronize());
    end_cpu = get_time();
    
    hipMemcpy(y_sw, c_y, num_rows * sizeof(float), hipMemcpyDeviceToHost);

    printf("\nSPMV Time GPU: %.10lf", end_cpu - start_cpu);

    hipFree(ptr);
    hipFree(indices);
    hipFree(data);
    hipFree(c_x);
    hipFree(c_y);

    // Free    
    free(row_ptr);
    free(col_ind);
    free(values);
    free(y_sw);
    
    return 0;
}